#include "hip/hip_runtime.h"
#include <cstdio>
#include <iostream>

#include "util/cuda-util.cuh"

template<typename T>
__global__ void write_kernel(T *device_ptr) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    device_ptr[i] = static_cast<T>(i);
}

template<typename T>
__global__ void read_kernel(T *device_ptr) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Index: %llu, Value: %d\n", i, static_cast<int>(device_ptr[i]));
}

void checkPointerType(void *ptr) {
    hipPointerAttribute_t attr{};
    hipError_t err = hipPointerGetAttributes(&attr, ptr);

    if (err != hipSuccess) {
        std::cout << "Error or not a CUDA-registered pointer: " << hipGetErrorString(err) << std::endl;
        return;
    }

    switch (attr.type) {
        case cudaMemoryTypeUnregistered:
            std::cout << "Unregistered / Unknown Memory (possibly regular CPU memory)\n";
            break;
        case hipMemoryTypeHost:
            std::cout << "Host Memory: Pinned or Mapped/Zero-copy (allocated via hipHostAlloc)\n";
            break;
        case hipMemoryTypeDevice:
            std::cout << "Device memory (allocated via hipMalloc)\n";
            break;
        case hipMemoryTypeManaged:
            std::cout << "Unified Managed Memory (allocated via hipMallocManaged)\n";
            break;
        default:
            std::cout << "Unknown memory type\n";
            break;
    }
}

template<typename T>
void test_mem(T *ptr, size_t size) {
    std::cout << "test_mem:" << ptr << "\n";
    checkPointerType(ptr);

    write_kernel<<<1, size>>>(ptr);
    read_kernel<<<1, size>>>(ptr);
    cuda_check(hipDeviceSynchronize());
    std::cout << "\n";
}

int main() {
    using dtype = float;
    size_t size = 16;
    size_t nBytes = size * sizeof(dtype);

    dtype *p1, *p2, *p3;

    // normal device memory
    cuda_check(hipMalloc(&p1, nBytes));

    // mapped memory
    cuda_check(hipHostAlloc(&p2, nBytes, hipHostMallocMapped));

    // mannaged memory
    cuda_check(hipMallocManaged(&p3, nBytes));

    // test
    test_mem<dtype>(p1, size);
    test_mem<dtype>(p2, size);
    test_mem<dtype>(p3, size);

    // free
    cuda_check(hipFree(p1));
    cuda_check(hipHostFree(p2));
    cuda_check(hipFree(p3));
}
