
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(int id) {
    printf("Kernel %d running on thread %d\n", id, threadIdx.x);
}

int main() {
    // create streams
    hipStream_t stream, stream2, stream3;
    hipStreamCreate(&stream);
    hipStreamCreate(&stream2);
    hipStreamCreate(&stream3);

    // create events
    hipEvent_t evt1, evt2, evt3;
    hipEventCreate(&evt1);
    hipEventCreate(&evt2);
    hipEventCreate(&evt3);

    // create graph
    hipGraph_t graph;
    hipGraphExec_t graphExec;

    // start stream capture
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    // --- Stage 1 ---
    kernel<<<1,1,0,stream>>>(1);
    hipEventRecord(evt1, stream);

    // --- Stage 2a ---
    hipStreamWaitEvent(stream2, evt1, 0);
    kernel<<<1,1,0,stream2>>>(2);
    hipEventRecord(evt2, stream2);

    // --- Stage 2b ---
    hipStreamWaitEvent(stream3, evt1, 0);
    kernel<<<1,1,0,stream3>>>(3);
    hipEventRecord(evt3, stream3);

    // --- Stage 3 ---
    hipStreamWaitEvent(stream, evt2, 0);
    hipStreamWaitEvent(stream, evt3, 0);
    kernel<<<1,1,0,stream>>>(4);

    // stop capture
    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0);

    // launch graph
    hipGraphLaunch(graphExec, stream);
    hipStreamSynchronize(stream);

    // cleanup
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);

    hipEventDestroy(evt1);
    hipEventDestroy(evt2);
    hipEventDestroy(evt3);

    hipStreamDestroy(stream);
    hipStreamDestroy(stream2);
    hipStreamDestroy(stream3);
}
