#include "hip/hip_runtime.h"
#include <cstdio>

#include "util/cuda-util.cuh"

__global__ void hello_from_kernel(size_t kernel_id) {
    printf(
        "Kernel %llu: Hello from (Block:%d, Thread:%d)\n",
        kernel_id, blockIdx.x, threadIdx.x
    );
}

void single_stream(size_t n_kernel) {
    std::cout << "Run with single stream:\n";

    hipStream_t stream;
    cuda_check(hipStreamCreate(&stream));

    for (int i = 0; i < n_kernel; i++) {
        hello_from_kernel<<<1, 1, 0, stream>>>(i);
    }

    cuda_check(hipStreamSynchronize(stream));
    cuda_check(hipStreamDestroy(stream));

    std::cout << "\n";
}

void multi_stream(size_t n_kernel) {
    std::cout << "Run with multi stream:\n";

    std::vector<hipStream_t> streams(n_kernel);
    for (auto &stream: streams) {
        cuda_check(hipStreamCreate(&stream));
    }

    for (int i = 0; i < n_kernel; i++) {
        hello_from_kernel<<<1, 1, 0, streams[i]>>>(i);
    }

    for (auto &stream: streams) {
        cuda_check(hipStreamSynchronize(stream));
    }

    for (auto &stream: streams) {
        cuda_check(hipStreamDestroy(stream));
    }

    std::cout << "\n";
}


int main() {
    size_t n_kernel = 100;
    single_stream(n_kernel);
    multi_stream(n_kernel);
}
